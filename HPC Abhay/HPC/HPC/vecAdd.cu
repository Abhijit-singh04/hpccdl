
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>

// CUDA kernel for vector addition
__global__ void vectorAdd(float *a, float *b, float *c, int n) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < n) {
        c[index] = a[index] + b[index];
    }
}

int main() {
    int n = 1000000; // Number of elements in the vectors
    size_t bytes = n * sizeof(float);

    // Host vectors
    float *h_a, *h_b, *h_c;
    h_a = (float*)malloc(bytes);
    h_b = (float*)malloc(bytes);
    h_c = (float*)malloc(bytes);

    // Initialize host vectors
    for (int i = 0; i < n; ++i) {
        h_a[i] = i;
        h_b[i] = 2 * i;
    }

    // Device vectors
    float *d_a, *d_b, *d_c;
    hipMalloc(&d_a, bytes);
    hipMalloc(&d_b, bytes);
    hipMalloc(&d_c, bytes);

    // Copy host vectors to device
    hipMemcpy(d_a, h_a, bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, bytes, hipMemcpyHostToDevice);

    // Launch kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;
    vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_a, d_b, d_c, n);

    // Copy result back to host
    hipMemcpy(h_c, d_c, bytes, hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    // Output result
    for (int i = 0; i < 10; ++i) {
        std::cout << h_c[i] << " ";
    }
    std::cout << std::endl;

    // Free host memory
    free(h_a);
    free(h_b);
    free(h_c);

    return 0;
}
